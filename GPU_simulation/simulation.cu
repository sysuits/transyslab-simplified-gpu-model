#include "hip/hip_runtime.h"

#include <cmath>
#include <stdint.h>
#include <fstream>
#include <sstream>
#include <iomanip>
#include <string>
#include <map>
#include "GPUmemory.h"
#include "simuResult.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstring>
#include <vector>
#include "GPUvehicle.h"
//#include "GPU_kernel.cuh"
#include "hip/device_functions.h"
#include "demand.h"
//#include <windows.h>
#include<time.h>
#include <assert.h>

using namespace std;

GPUmemory* gpu_data;
GPUvehicle *CPU_veh;
GPUvehicle *GPU_veh;
Demand *gpu_demand;
int memory_space_for_vehicles = TotalVehNum * sizeof(GPUvehicle);

simuResult* GPUResult;
simuResult* CPUResult=NULL;

bool InitilizeGPU(string road_file, string lane_file, string veh_file, string node_file);     
bool InitGPUData(GPUmemory* cpu_data,Demand* cpu_demand,string road_file, string lane_file, string veh_file, string node_file);            //��ʼ�� GPUmemory
bool StartSimulation();

vector<string> &split(const std::string &s, char delim, std::vector<std::string> &elems)
{
	std::stringstream ss(s);
	std::string item;
	while (std::getline(ss, item, delim)) {
		elems.push_back(item);
	}
	return elems;
}
vector<string> split_str(string str, string pattern)
{
	string::size_type pos;
	vector<string> result;
	str += pattern;
	size_t size = str.size();
	for (size_t i = 0; i < size; i++)
	{
		pos = str.find(pattern, i);
		if (pos < size)
		{
			string s = str.substr(i, pos - i);
			result.push_back(s);
			i = pos + pattern.size() - 1;
		}
	}
	return result;
}

//���������
int main(int argc, char** argv) {
	printf("argc ->%d\n", argc);
	for (int i = 0; i < argc; i++)
	{
		std::cout << "argument[" << i << "] is: " << argv[i] << endl;
	}
	string road_file = argv[1];
	string lane_file = argv[2];
	string veh_file = argv[3];
	string node_file = argv[4];
	//string output_file= argv[5];
	//string MIT_node_file= argv[6];
	InitilizeGPU(road_file, lane_file, veh_file, node_file); //��ʼ��CPU��GPU���ݲ�����
	std::cout << "Initlition Complete" << std::endl;
	StartSimulation();   //���л�����
	std::cout << "Simulation Succeed!" << endl;
	//hipDeviceReset();   //����GPU��Ϣ
}

bool InitilizeGPU(string road_file, string lane_file, string veh_file, string node_file) {
	gpu_data = NULL;
	GPUmemory* cpu_data = new GPUmemory();
	Demand* cpu_demand = new Demand();
	InitGPUData(cpu_data, cpu_demand, road_file, lane_file, veh_file, node_file); //��ʼ����������
	//���ݴ���
	if (hipMalloc((void**)&GPU_veh, memory_space_for_vehicles) != hipSuccess) {
		cerr << "hipMalloc((void**)&GPU_veh, memory_space_for_vehicles" << endl;
	}
	if (hipMalloc((void**)&gpu_demand, sizeof(NewVehicle)*TotalTimeSteps) != hipSuccess) {
		cerr<<"hipMalloc((void**)&gpu_demand, sizeof(NewVehicle)*TotalTimeSteps) failed"<< endl;
	}
	if (hipMalloc((void**)&gpu_data, (cpu_data->total_size())) != hipSuccess) {
		cerr << "hipMalloc((void**)&gpu_data, 2 * (cpu_data->total_size()) failed" << endl;
	}
	if (hipMalloc((void**)&GPUResult, sizeof(simuResult)) != hipSuccess){
		cerr << "hipMalloc((void**)&GPUResult, memory_space_outputs) failed" << endl;
	}
	if (hipHostMalloc((void **)&CPUResult, sizeof(simuResult)) != hipSuccess){
		cerr <<"hipHostMalloc((void **)&CPUResult, sizeof(simuResult)) failed"<< endl;
	}
	//hipMemcpy(GPUResult, CPUResult, memory_space_outputs, hipMemcpyHostToDevice);
	if (hipMemcpy(GPU_veh, CPU_veh, memory_space_for_vehicles, hipMemcpyHostToDevice) != hipSuccess) {
		cerr << "hipMemcpy(GPU_veh, CPU_veh, memory_space_for_vehicles, hipMemcpyHostToDevice failed" << endl;
	}
	if (hipMemcpy(gpu_data, cpu_data, (cpu_data->total_size()), hipMemcpyHostToDevice)!= hipSuccess) {
		cerr << "hipMemcpy(gpu_data, cpu_data, (cpu_data->total_size()), hipMemcpyHostToDevice); failed" << endl;
	}
	if (hipMemcpy(gpu_demand, cpu_demand, sizeof(NewVehicle)*TotalTimeSteps, hipMemcpyHostToDevice) != hipSuccess) {
		cerr << "hipMemcpy(gpu_demand, cpu_demand, sizeof(NewVehicle)*TotalTimeSteps, hipMemcpyHostToDevice) failed" << endl;
	}
	return true;
}

//��ʼ����������
bool InitGPUData(GPUmemory* cpu_data, Demand* cpu_demand,string road_file, string lane_file, string veh_file, string node_file) {
	
	int veh_start = 0;
	int veh_index = 0;
	//���������ļ�
	for (int q = 0; q < TotalTimeSteps; q++) {
		for (int p = 0; p < RoadSize; p++) {
			cpu_demand->new_vehicle_everystep[q].Lane_ID[p] = p;
			cpu_demand->new_vehicle_everystep[q].new_vehicle_size[p] = 0;
		}
	}
	for (int i = 0; i < NetworkMaxNum; i++) {
		cpu_data->lane_vehicle_space.vehicle_space[i] = -1;
		cpu_data->buffer_vehicle_space.buffer_space[i] = -1;
	}
	std::cout << "new_vehicle_everystep initial complete" << endl;
	//����·���ļ�
	std::string line;
	std::ifstream myfile(road_file.c_str());
	vector<int> road_len;
	if (myfile.is_open()) {
		int index = 0;
		while (getline(myfile, line)) {
			vector<string> res;
			split(line, ':', res);
			cpu_data->road_pool.Road_ID[index] = atoi(res[0].c_str());
			cpu_data->road_pool.up_node_index[index] = atoi(res[1].c_str());
			cpu_data->road_pool.down_node_index[index] = atoi(res[2].c_str());
			cpu_data->road_pool.length[index] = atoi(res[3].c_str());
			cpu_data->road_pool.lane_start_index[index] = atoi(res[4].c_str());
			cpu_data->road_pool.lane_end_index[index] = atoi(res[5].c_str());
			cpu_data->road_pool.lane_count[index] = atoi(res[5].c_str()) - atoi(res[4].c_str());
			cpu_data->road_pool.park_num[index]=0;
			cpu_data->road_pool.park_output[index] = 2;
			road_len.push_back(atoi(res[3].c_str()));
			index++;
		}
		myfile.close();
	}
	std::cout << "road data input complete" << endl;
	//���س����ļ�
	std::string line2;
	std::ifstream myfile2(lane_file.c_str());
	int lane_num = 0;
	if (myfile2.is_open()) {
		int lane_index = 0;
		int lane_ID = 0;
		while (getline(myfile2, line2)) {
			vector<string> res;
			split(line2, ':', res);
			cpu_data->lane_pool.lane_ID[lane_index] = atoi(res[0].c_str());
			cpu_data->lane_pool.direction[lane_index] = atoi(res[1].c_str());
			cpu_data->lane_pool.road_ID[lane_index] = atoi(res[9].c_str());
			//std::cout << cpu_data->lane_pool.lane_ID[lane_index] << '-' << cpu_data->lane_pool.road_ID[lane_index] << endl;
			cpu_data->lane_pool.flow[lane_index] = 0;
			cpu_data->lane_pool.density[lane_index] = 0;
			cpu_data->lane_pool.speed[lane_index] = 0;
			cpu_data->lane_pool.queue_length[lane_index] = 0;
			cpu_data->lane_pool.lane_length[lane_index] = atoi(res[6].c_str());
			cpu_data->lane_pool.alpha[lane_index] = 1;
			cpu_data->lane_pool.beta[lane_index] = 1;
			cpu_data->lane_pool.max_density[lane_index] = 178.6;
			cpu_data->lane_pool.max_speed[lane_index] = 60;
			cpu_data->lane_pool.min_speed[lane_index] = 10.8;
			cpu_data->lane_pool.vehicle_counts[lane_index] = 0;
			cpu_data->lane_pool.vehicle_start_index[lane_index] = atoi(res[7].c_str());
			cpu_data->lane_pool.vehicle_end_index[lane_index] = atoi(res[8].c_str());
			int max_veh = cpu_data->lane_pool.vehicle_end_index[lane_index];
			int min_veh= cpu_data->lane_pool.vehicle_start_index[lane_index];
			cpu_data->lane_pool.max_vehicles[lane_index] = max_veh-min_veh;
			cpu_data->lane_pool.output_capacity[lane_index] = 2;
			cpu_data->lane_pool.empty_space[lane_index] = max_veh - min_veh;
			//printf("empty: %d \n", cpu_data->lane_pool.empty_space[lane_index]);
			cpu_data->lane_pool.buffer_counts[lane_index] = 0;
			cpu_data->lane_pool.buffered_vehicle_start_index[lane_index] = atoi(res[7].c_str());
			cpu_data->lane_pool.buffered_vehicle_end_index[lane_index] = atoi(res[8].c_str());
			cpu_data->lane_pool.vehicle_passed[lane_index] = false;
			cpu_data->lane_pool.signal[lane_index] = atoi(res[2].c_str());
			cpu_data->lane_pool.greenstart_time[lane_index] = atoi(res[3].c_str());
			cpu_data->lane_pool.green[lane_index] = atoi(res[4].c_str());
			cpu_data->lane_pool.cycle_offset[lane_index] = atoi(res[5].c_str());
			cpu_data->lane_pool.complete_num[lane_index] = 0;
			cpu_data->lane_pool.begin_num[lane_index] = 0;
			//cpu_data->lane_pool.locked[lane_index] = false;
			lane_index++;
			lane_ID++;
			lane_num++;
		}
		myfile2.close();
	}
	std::cout << "lane data input complete" << endl;
	//���س����ļ�
	std::string line3;
	std::ifstream myfile3(veh_file.c_str());
	CPU_veh = (GPUvehicle*)malloc(memory_space_for_vehicles);
	//memset(CPU_veh, 0, memory_space_for_vehicles);
	int veh_num = 0;
	if (myfile3.is_open()) {
		int veh_index = 0;
		int veh_ID =0;
		while (getline(myfile3, line3)) {
			veh_num += 1;
			vector<string> res;
			split(line3, ':', res);
			CPU_veh[veh_index].vehicle_ID = veh_ID;
			int entry_time = atoi(res[1].c_str());
			CPU_veh[veh_index].entry_time = atoi(res[1].c_str());
			string::size_type idx;
			idx = res[2].find('-');
			if (idx == string::npos) {
				CPU_veh[veh_index].path_road[0] = atoi(res[2].c_str());
				CPU_veh[veh_index].path_directions[0] = atoi(res[3].c_str());
				CPU_veh[veh_index].whole_num = 1;
			}
			else {
				vector<string> road_path = split_str(res[2], "-");
				vector<string> fx_path = split_str(res[3], "-");
				for (int j = 0; j < road_path.size(); j++) {
					CPU_veh[veh_index].path_road[j] = atoi(road_path[j].c_str());
					CPU_veh[veh_index].path_directions[j] = atoi(fx_path[j].c_str());
				}
				CPU_veh[veh_index].whole_num = road_path.size();
			}
			CPU_veh[veh_index].path_num = 0;
			int st = CPU_veh[veh_index].path_road[0];
			CPU_veh[veh_index].current_road_ID = st;
			CPU_veh[veh_index].current_lane_ID = -1;
			CPU_veh[veh_index].distant = 100;
			CPU_veh[veh_index].next_lane = -1;
			CPU_veh[veh_index].next_road = -1;
			int road_index = CPU_veh[veh_index].path_road[0];
			if (cpu_demand->new_vehicle_everystep[entry_time].new_vehicle_size[road_index] < LaneInputCapacity) {
				int insert_index = cpu_demand->new_vehicle_everystep[entry_time].new_vehicle_size[road_index];
				cpu_demand->new_vehicle_everystep[entry_time].new_vehicles[road_index][insert_index] = veh_index;
				cpu_demand->new_vehicle_everystep[entry_time].new_vehicle_size[road_index]++;
			}
			veh_index++;
			veh_ID++;
		}
		myfile3.close();
	}
	std::cout << "vehicle/demand data input complete" << endl;
	std::cout << "total vehicle num:"<<veh_num << endl;
	//���ؽڵ��ļ�
	std::string line4;
	std::ifstream myfile4(node_file.c_str());
	if (myfile4.is_open()) {
		while (getline(myfile4, line4)) {
			vector<string> res;
			split(line4, ':', res);
			//int index = 0;
			int index = atoi(res[0].c_str());
			cpu_data->node_pool.Node_ID[index] = index;
			cpu_data->node_pool.current_buffer[index] = index;
			cpu_data->node_pool.buffer_counts[index] = 0;
			string::size_type idx;
			idx = res[1].find('-');
			if (idx == string::npos) {
				if (res[1] == "null") {
					cpu_data->node_pool.up_lane_num[index] = 0;
				}
				else {
					int up = atoi(res[1].c_str());
					cpu_data->node_pool.up_lane_num[index] = 1;
					cpu_data->node_pool.up_lane_index[index][0] = up;
				}
			}
			else {
				vector<string> up_lane = split_str(res[1], "-");
				cpu_data->node_pool.up_lane_num[index] = up_lane.size();
				for (int j = 0; j < up_lane.size(); j++) {
					cpu_data->node_pool.up_lane_index[index][j] = atoi(up_lane[j].c_str());
				}
			}
		}
		myfile4.close();
	}
	std::cout << "node data input complete" << endl;
	return true;
}

//�˺���������Ѱ��Ŀ�공��
__device__ int Vehicle_getnextlane(GPUmemory* gpu_data, int road_index, int direction) {
	int lane_ID = -1;
	int lane_queue = -1;
	for (int i = gpu_data->road_pool.lane_start_index[road_index]; i < gpu_data->road_pool.lane_end_index[road_index]; i++) {
		int dir1 = gpu_data->lane_pool.direction[i];
		int dir2 = direction;
		if (gpu_data->lane_pool.empty_space[i] > 0) {  //�����Ƿ���ʣ������
			//ѡ��ת�򳵵�
		    if (dir1 == dir2 || (dir2 == 0 && (dir1 == 3 || dir1 == 5 || dir1 == 6)) || (dir2 == 1 && (dir1 == 4 || dir1 == 5 || dir1 == 6)) || (dir2 == 2 && (dir1 == 4 || dir1 == 3 || dir1 == 6))) {
				if (lane_queue < 0 ||((lane_queue >= 0) && (gpu_data->lane_pool.queue_length[i] < lane_queue))) {
					lane_ID = i;
					lane_queue = gpu_data->lane_pool.queue_length[i];
				}
			}
		}
	}
	if (lane_queue == -1 || lane_ID == -1) {
		return -1;
	}
	else {
		return lane_ID; //���Ŀ�공��
	}
}

//�˺�������·��Ϊ���е�Ԫ�����ء�����������
__global__ void load_demand(GPUmemory*gpu_data, Demand* gpu_demand, int time_step, int Roadsize, GPUvehicle * GPU_veh) {
	unsigned road_index = blockIdx.x * blockDim.x + threadIdx.x;
	if (road_index >= Roadsize)
		return;
	int flow = 0;
	//int length = gpu_data->road_pool.length[road_index];
	for (int i = gpu_data->road_pool.lane_start_index[road_index]; i < gpu_data->road_pool.lane_end_index[road_index]; i++) {
		flow+=gpu_data->lane_pool.flow[i];
	}
	//·���ܶȸ���
	float density = flow /((gpu_data->road_pool.lane_count[road_index] * gpu_data->road_pool.length[road_index]) / 1000);
	gpu_data->road_pool.density[road_index] =density;
	//·���ٶȸ���
	float speed = 10.8 + (60 -10.8)*(1 - (density /178.6));
	gpu_data->road_pool.speed[road_index] = speed / 3.6;
	//����������
	for (int i = 0; i < gpu_demand->new_vehicle_everystep[time_step].new_vehicle_size[road_index]; i++) {
		int veh = gpu_demand->new_vehicle_everystep[time_step].new_vehicles[road_index][i];
		int obj_lane = Vehicle_getnextlane(gpu_data,road_index, GPU_veh[veh].path_directions[0]); //�����󳵵�ѡ��
		if ( obj_lane!=-1) {
			//��������״̬��Ϣ����
			GPU_veh[veh].current_lane_ID = obj_lane;
			GPU_veh[veh].start_time = time_step;
			GPU_veh[veh].distant = gpu_data->lane_pool.lane_length[obj_lane];
			gpu_data->lane_pool.flow[obj_lane] += 1;
			gpu_data->lane_pool.empty_space[obj_lane] -= 1;
			//���س���
			int load_point = gpu_data->lane_pool.vehicle_start_index[obj_lane] + gpu_data->lane_pool.vehicle_counts[obj_lane];
			gpu_data->lane_vehicle_space.vehicle_space[load_point] = veh;
			gpu_data->lane_pool.vehicle_counts[obj_lane] += 1;
			gpu_data->lane_pool.begin_num[obj_lane] += 1;
			//printf("%d new---------------------obj:%d,veh:%d,road:%d,where:%d\n", time_step, obj_lane, veh, road_index,p);
		}
		else {
			//���������������ת����·��ͣ����
			//printf("transfer to park--------------------obj:%d,veh:%d,road:%d\n", obj_lane, veh, road_index);
			int n = gpu_data->road_pool.park_num[road_index];
			gpu_data->road_pool.park[road_index][n] = veh;
			gpu_data->road_pool.park_num[road_index]++;
		}
	}
	//·��ͣ�����ŷ�ʱ�����
	if (gpu_data->road_pool.park_output[road_index]>0) {
		gpu_data->road_pool.park_output[road_index]--;
	}
	if (gpu_data->road_pool.park_num[road_index] > 0) {
		if (gpu_data->road_pool.park_output[road_index]==0) {
			int veh = gpu_data->road_pool.park[road_index][0];//ͣ����������
			int obj_lane = Vehicle_getnextlane(gpu_data, road_index, GPU_veh[veh].path_directions[0]);
			//GPU_veh[veh].next_lane = obj_lane;
			if (obj_lane != -1) {
				//ͣ�����ŷų���
				GPU_veh[veh].current_lane_ID = gpu_data->lane_pool.lane_ID[obj_lane];
				GPU_veh[veh].start_time = time_step;
				GPU_veh[veh].distant == gpu_data->lane_pool.lane_length[obj_lane];
				//printf("%d park out ---------------------obj:%d,veh:%d,road:%d\n", time_step, obj_lane, veh, road_index);
				gpu_data->lane_pool.flow[obj_lane] += 1;
				gpu_data->lane_pool.empty_space[obj_lane] -= 1;
				int p = gpu_data->lane_pool.vehicle_start_index[obj_lane] + gpu_data->lane_pool.vehicle_counts[obj_lane];
				gpu_data->lane_vehicle_space.vehicle_space[p] = veh;
				gpu_data->lane_pool.vehicle_counts[obj_lane] += 1;
				for (int j = 0; j < gpu_data->road_pool.park_num[road_index];j++) {
					gpu_data->road_pool.park[road_index][j] = gpu_data->road_pool.park[road_index][j+1];
				}
				gpu_data->road_pool.park[road_index][gpu_data->road_pool.park_num[road_index]] = 0;
				gpu_data->road_pool.park_num[road_index] -= 1;
				gpu_data->road_pool.park_output[road_index] = 2;
				gpu_data->lane_pool.begin_num[obj_lane] += 1;
				gpu_data->road_pool.begin_num[road_index] += 1;
			}
		}
	}
	return;
}

//�˺������Գ���Ϊ���е�Ԫ�������ӻ������·ţ�����������״̬����
__global__ void simulateVehicle_pass(GPUmemory*gpu_data, int time_step, int LaneSize, GPUvehicle * GPU_veh, simuResult* GPUResult) {
	unsigned lane_index = blockIdx.x * blockDim.x + threadIdx.x;
	if (lane_index >= LaneSize)
		return;
	//�����ٶȸ��£�=����·���ٶȣ�
	float speed = gpu_data->road_pool.speed[gpu_data->lane_pool.road_ID[lane_index]];  
	//�����ӳ����������·�����ʵ����
	for (int i = 0; i < gpu_data->node_pool.buffer_counts[lane_index]; i++) {
		//����״̬����
		int veh_id = gpu_data->buffer_vehicle_space.buffer_space[gpu_data->lane_pool.buffered_vehicle_start_index[lane_index] + i];
		//printf("load in real lane,veh:%d,lane:%d,where:%d\n", veh_id, lane_index, gpu_data->lane_pool.buffered_vehicle_start_index[lane_index] + i);
		GPU_veh[veh_id].current_lane_ID = lane_index;
		GPU_veh[veh_id].entry_time = time_step;
		GPU_veh[veh_id].distant = gpu_data->lane_pool.lane_length[lane_index];
		GPU_veh[veh_id].path_num += 1;
		gpu_data->lane_vehicle_space.vehicle_space[gpu_data->lane_pool.vehicle_counts[lane_index] + gpu_data->lane_pool.vehicle_start_index[lane_index]] = veh_id;
		gpu_data->lane_pool.vehicle_counts[lane_index] += 1;
		gpu_data->lane_pool.flow[lane_index] += 1;
		gpu_data->buffer_vehicle_space.buffer_space[gpu_data->lane_pool.buffered_vehicle_start_index[lane_index] + i] = 0;
		//gpu_data->node_pool.buffer_counts[lane_index] -= 1;

	}
    gpu_data->node_pool.buffer_counts[lane_index] = 0;
	//�ſظ���
	//gpu_data->lane_pool.signal[lane_index] = -1;
	if (gpu_data->lane_pool.signal[lane_index] != -1) { //Ϊ�źſ��Ƶĳ���
		int temp = time_step - gpu_data->lane_pool.greenstart_time[lane_index];
		if (temp > gpu_data->lane_pool.cycle_offset[lane_index]) {
			gpu_data->lane_pool.greenstart_time[lane_index] += gpu_data->lane_pool.cycle_offset[lane_index];
		}
		if (temp>=0 && temp <=gpu_data->lane_pool.green[lane_index]) {
			gpu_data->lane_pool.signal[lane_index] = 1;
		}
		else {
			gpu_data->lane_pool.signal[lane_index] = 0;
		}
	}
	//�����������
	for (int i = gpu_data->lane_pool.vehicle_start_index[lane_index]; i < gpu_data->lane_pool.vehicle_start_index[lane_index] + gpu_data->lane_pool.vehicle_counts[lane_index]; i++) {
		if (GPU_veh[gpu_data->lane_vehicle_space.vehicle_space[i]].distant > 0) {
			GPU_veh[gpu_data->lane_vehicle_space.vehicle_space[i]].distant -= speed;
			if (GPU_veh[gpu_data->lane_vehicle_space.vehicle_space[i]].distant <= 0) {
				gpu_data->lane_pool.queue_length[lane_index] += 1;  //���������Ŷ�
				gpu_data->lane_pool.flow[lane_index] -= 1;
			}
		}
	}
	//�����Ƿ�������ŷ�������ͨ���������̵ơ����Ŷӳ���
	if (gpu_data->lane_pool.queue_length[lane_index] > 0 && gpu_data->lane_pool.output_capacity[lane_index] == 0 && (gpu_data->lane_pool.signal[lane_index] == 1 || gpu_data->lane_pool.signal[lane_index] == -1)) {
		gpu_data->lane_pool.vehicle_passed[lane_index] = true;
	}
	else {
		gpu_data->lane_pool.vehicle_passed[lane_index] = false;
	}
	int start = gpu_data->lane_pool.vehicle_start_index[lane_index];
	int first_veh = gpu_data->lane_vehicle_space.vehicle_space[start];

	//ÿ���5minͳ�Ƴ�����Ϣ
	if (time_step % 300 == 0 && time_step > 0) {
		//printf("time:%d lane:%d speed:%d first_veh:%d count:%d signal:%d can_go:%d queue:%d,travel:%d\n", time_step, lane_index, speed, gpu_data->lane_vehicle_space.vehicle_space[gpu_data->lane_pool.vehicle_start_index[lane_index]], gpu_data->lane_pool.vehicle_counts[lane_index], gpu_data->lane_pool.signal[lane_index], gpu_data->lane_pool.vehicle_passed[lane_index], gpu_data->lane_pool.queue_length[lane_index], GPUResult->travel[lane_index]);
		int t = time_step / 300 - 1;
		GPUResult->flow[t][lane_index] = gpu_data->lane_pool.flow[lane_index];
		GPUResult->count[t][lane_index] = gpu_data->lane_pool.vehicle_counts[lane_index];
		GPUResult->speed[t][lane_index] = gpu_data->road_pool.speed[gpu_data->lane_pool.road_ID[lane_index]];
		//GPUResult->signal[t][lane_index] = gpu_data->lane_pool.signal[lane_index];
		//printf("time:%d,lane:%d,count:%d,travel:%d\n",time_step,lane_index, gpu_data->lane_pool.vehicle_counts[lane_index], GPUResult->travel[lane_index]);
		if (GPUResult->comlete_num[lane_index] > 0) {
			GPUResult->avg_travel[t][lane_index] = GPUResult->travel[lane_index] / GPUResult->comlete_num[lane_index];
			GPUResult->complete[t][lane_index] = GPUResult->comlete_num[lane_index];
			GPUResult->travel[lane_index] = 0;
			GPUResult->comlete_num[lane_index] = 0;
			//gpu_data->lane_pool.complete_num[lane_index] = 0;
		}
	}

	gpu_data->lane_pool.empty_space[lane_index] = gpu_data->lane_pool.max_vehicles[lane_index]-gpu_data->lane_pool.vehicle_counts[lane_index];
	printf("time:%d lane:%d first_veh:%d distant:%f count:%d empty:%d road:%d can_go:%d queue:%d,last_veh:%d\n", time_step, lane_index ,gpu_data->lane_vehicle_space.vehicle_space[gpu_data->lane_pool.vehicle_start_index[lane_index]], GPU_veh[first_veh].distant, gpu_data->lane_pool.vehicle_counts[lane_index], gpu_data->lane_pool.empty_space[lane_index], gpu_data->lane_pool.road_ID[lane_index], gpu_data->lane_pool.vehicle_passed[lane_index], gpu_data->lane_pool.queue_length[lane_index], gpu_data->lane_vehicle_space.vehicle_space[gpu_data->lane_pool.vehicle_start_index[lane_index] + gpu_data->lane_pool.vehicle_counts[lane_index] - 1]);
	if (gpu_data->lane_pool.vehicle_passed[lane_index] = true && gpu_data->lane_pool.queue_length[lane_index] > 0 && GPU_veh[first_veh].distant <= 0) {
		if (GPU_veh[first_veh].path_num == GPU_veh[first_veh].whole_num - 1) {   //�ó����г̽���
			gpu_data->lane_pool.complete_num[lane_index] += 1;
			GPUResult->comlete_num[lane_index] += 1;
			GPUResult->total_complete[lane_index] += 1;
			GPU_veh[first_veh].com_time = time_step;
			GPUResult->travel[lane_index]+=time_step- GPU_veh[first_veh].entry_time;
			gpu_data->lane_vehicle_space.vehicle_space[start] = -1;
			for (int j = 0; j < gpu_data->lane_pool.vehicle_counts[lane_index] - 1; j++) {
				gpu_data->lane_vehicle_space.vehicle_space[start + j] = gpu_data->lane_vehicle_space.vehicle_space[start + j + 1];  //���γ���������ǰ��λ
			}
			gpu_data->lane_pool.empty_space[lane_index] += 1;
			gpu_data->lane_pool.vehicle_counts[lane_index] -= 1;
			gpu_data->lane_pool.queue_length[lane_index]-= 1;
		}
		else {  //�ó��������г�
			int current = GPU_veh[first_veh].path_num + 1;
			int obj_lane = Vehicle_getnextlane(gpu_data, GPU_veh[first_veh].path_road[current], GPU_veh[first_veh].path_directions[current]);
			GPU_veh[first_veh].next_lane = obj_lane; //ѡ������Ŀ�공��
			if (obj_lane == -1) {
				gpu_data->lane_pool.vehicle_passed[lane_index] = false;  //���׳��޷��ŷţ���������
			}
		}
	}
	//�����ŷ�ʱ�����
	if (gpu_data->lane_pool.output_capacity[lane_index] > 0) {
		gpu_data->lane_pool.output_capacity[lane_index] -= 1;
	}
	///__syncthreads();
	GPUResult->begin_num[lane_index] = gpu_data->lane_pool.begin_num[lane_index];
	//GPUResult->total_complete[lane_index] = gpu_data->lane_pool.complete_num[lane_index];
	GPUResult->comlete_num[lane_index] = gpu_data->lane_pool.complete_num[lane_index];
	return;
}

//�˺������Գ���������-�ڵ�Ϊ���е�Ԫ�����������γ���ת��������Ŀ�공��������
__global__ void simulateVehicle_prepass(GPUmemory*gpu_data, int time_step, int LaneSize, GPUvehicle * GPU_veh, simuResult* GPUResult) {
	unsigned node_index = blockIdx.x * blockDim.x + threadIdx.x;
	if (node_index >= LaneSize)
		return;
	//�����ڵ����ι�������
	for (int i = 0; i < gpu_data->node_pool.up_lane_num[node_index]; i++) {
		int up_lane = gpu_data->node_pool.up_lane_index[node_index][i]; //���γ�����
		int first_up_veh = gpu_data->lane_vehicle_space.vehicle_space[gpu_data->lane_pool.vehicle_start_index[up_lane]];
		//�����γ���ͷ��Ŀ�공��=�ó���������
		if (GPU_veh[first_up_veh].next_lane == gpu_data->lane_pool.lane_ID[node_index]) {
			//�����γ��������ŷŵ�����
			if ((gpu_data->lane_pool.queue_length[up_lane]>0) && (gpu_data->lane_pool.output_capacity[up_lane]==0) && (gpu_data->lane_pool.vehicle_passed[up_lane] = true) && (gpu_data->lane_pool.empty_space[node_index] > 0)) {
				int buffer_insert = gpu_data->lane_pool.buffered_vehicle_start_index[node_index] + gpu_data->node_pool.buffer_counts[node_index]; //+ gpu_data->lane_pool.buffer_counts[node_index];
				gpu_data->buffer_vehicle_space.buffer_space[buffer_insert] = first_up_veh;    //ת�������λ�����
				GPUResult->travel[up_lane] += time_step - GPU_veh[first_up_veh].entry_time;
				GPUResult->comlete_num[up_lane] += 1;
				int start = gpu_data->lane_pool.vehicle_start_index[up_lane];
				gpu_data->lane_vehicle_space.vehicle_space[start] = -1;
				for (int j = 0; j < gpu_data->lane_pool.vehicle_counts[up_lane] - 1; j++) {
					gpu_data->lane_vehicle_space.vehicle_space[start + j] = gpu_data->lane_vehicle_space.vehicle_space[start + j + 1];  //���γ���������ǰ��λ
				}
				gpu_data->lane_pool.vehicle_counts[up_lane] -= 1;
				gpu_data->lane_pool.queue_length[up_lane] -= 1;      //����������
				gpu_data->lane_pool.empty_space[up_lane] += 1;
				gpu_data->lane_pool.empty_space[node_index] -= 1;    //������������
				gpu_data->lane_pool.output_capacity[up_lane] = 2;  //ͨ��������λ
				gpu_data->node_pool.buffer_counts[node_index] += 1;
			}
		}
	}
	return;
}


__global__ void result_toCPU(GPUmemory* gpu_data, int LaneSize, simuResult* GPUResult) {
	int lane_index = blockIdx.x * blockDim.x + threadIdx.x;
	if (lane_index >= LaneSize)
		return;
}

//CUDA�쳣��׽����
#define CUDA_ERROR_CHECK
#define CudaSafeCall( err ) __cudaSafeCall( err, __FILE__, __LINE__ )
#define CudaCheckError()    __cudaCheckError( __FILE__, __LINE__ )
inline void __cudaSafeCall(hipError_t err, const char *file, const int line)
{
#ifdef CUDA_ERROR_CHECK
	if (hipSuccess != err)
	{
		fprintf(stderr, "cudaSafeCall() failed at %s:%i : %s\n",
			file, line, hipGetErrorString(err));
		exit(-1);
	}
#endif

	return;
}
inline void __cudaCheckError(const char *file, const int line)
{
#ifdef CUDA_ERROR_CHECK
	hipError_t err = hipGetLastError();
	if (hipSuccess != err)
	{
		fprintf(stderr, "cudaCheckError() failed at %s:%i : %s\n",
			file, line, hipGetErrorString(err));
		exit(-1);
	}
	err = hipDeviceSynchronize();
	if (hipSuccess != err)
	{
		fprintf(stderr, "cudaCheckError() with sync failed at %s:%i : %s\n",
			file, line, hipGetErrorString(err));
		exit(-1);
	}
#endif

	return;
}

//���з���������
bool StartSimulation() {
	hipStream_t gpu_stream;
	hipStreamCreate(&gpu_stream);
	unsigned thread = 128;
	unsigned road_block = ceil(1.0f *RoadSize / thread);
	unsigned lane_block = ceil(1.0f *LaneSize / thread);
	int time_step = 1;   //������
	int end_time =1000;  //�������ʱ��
	
	hipEvent_t start,stop;//�¼�����  
	hipEventCreate(&start);//�����¼�  
	hipEventCreate(&stop);//�����¼�  
	hipEventRecord(start,Stream0);//��¼��ʼ  

	for (int now_time = 0; now_time < end_time; now_time++) {
		printf("%d\n", now_time);
		load_demand<<<road_block, thread, 0, gpu_stream >>> (gpu_data, gpu_demand, now_time, RoadSize, GPU_veh);
		simulateVehicle_pass <<<lane_block, thread, 0, gpu_stream >>> (gpu_data, now_time, LaneSize, GPU_veh, GPUResult);
		simulateVehicle_prepass <<<lane_block, thread, 0, gpu_stream >>> (gpu_data, now_time, LaneSize, GPU_veh, GPUResult);
		//hipMemcpy(CPUResult, GPUResult, sizeof(simuResult), hipMemcpyDeviceToHost);  //���ݽ��
		//hipDeviceSynchronize();
		CudaCheckError();
	    //result_toCPU <<<block, thread, 0, gpu_stream >>> (gpu_data, LaneSize, GPUResult);
	}
	hipMemcpy(CPUResult, GPUResult, sizeof(simuResult), hipMemcpyDeviceToHost);  //����GPU��������CPU
	
	hipEventRecord(stop,Stream0);//��¼�����¼�  
	hipEventSynchronize(stop);//�¼�ͬ�����ȴ������¼�֮ǰ���豸�����������  
	float elapsedTime;  
	hipEventElapsedTime(&elapsedTime,start,stop);//���������¼�֮��ʱ������λΪms��  
	
	printf("GPU Elapsed time:%.6f ms.\n",elapsedTime);  

	
	std::cout << "END =============================================" << endl;
	int p = 0;
	int q = 0;
	for (int i = 0; i < LaneSize; i++) {
		p += CPUResult->total_complete[i];
		q += CPUResult->begin_num[i];
	}
	std::cout << "complete" << p << endl;  //������г̳�����
	std::cout << "generate" << q << endl;  //�ܼ���������
	
	ofstream out("lane_output.txt"); //��������ͳ�ƽ����Ŀ���ļ�
	for (int j = 0; j < 12; j++)
	{
		for (int i = 0; i < LaneSize; i++) {
			out << "time_step:" << j + 1 << ",lane:" << i << ",count:" << CPUResult->count[j][i] << ",speed:" << CPUResult->speed[j][i] << ",travel:" << CPUResult->avg_travel[j][i] <<",complete:"<< CPUResult->complete[j][i]<<",flow:"<< CPUResult->flow[j][i] << "\n";
		}
	}
	out.close();
	hipDeviceReset();   //����GPU��Ϣ
	return true;
}
